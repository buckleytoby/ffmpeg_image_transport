#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2016 Avionic Design GmbH
 * Meike Vocke <meike.vocke@avionic-design.de>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License version 2 as
 * published by the Free Software Foundation.
 *
 * This file contains the CUDA kernel with functions to initialise all needed
 * parameters for kernel Launch. Also the destruction of generated parameter is
 * included.
 *
 * Compute Capability 3.0 or higher required
 */

// from here: https://github.com/avionic-design/cuda-debayer/blob/master/src/bayer2rgb.cu

#include "ffmpeg_image_transport/cuda_encoder.hpp"

#define LEFT(x, y, imgw)	((x) - 1 + (y) * (imgw))
#define RIGHT(x, y, imgw)	((x) + 1 + (y) * (imgw))
#define TOP(x, y, imgw)		((x) + ((y) - 1) * (imgw))
#define BOT(x, y, imgw)		((x) + ((y) + 1) * (imgw))
#define TL(x, y, imgw)		((x) - 1 + ((y) - 1) * (imgw))
#define BL(x, y, imgw)		((x) - 1 + ((y) + 1) * (imgw))
#define TR(x, y, imgw)		((x) + 1 + ((y) - 1) * (imgw))
#define BR(x, y, imgw)		((x) + 1 + ((y) + 1) * (imgw))

#define PIX(in, x, y, imgw) \
	in[((x) + (y) * (imgw))]

#define INTERPOLATE_H(in, x, y, w) \
	(((uint32_t)in[LEFT(x, y, w)] + in[RIGHT(x, y, w)]) / 2)

#define INTERPOLATE_V(in, x, y, w) \
	(((uint32_t)in[TOP(x, y, w)] + in[BOT(x, y, w)]) / 2)

#define INTERPOLATE_HV(in, x, y, w) \
	(((uint32_t)in[LEFT(x, y, w)] + in[RIGHT(x, y, w)] + \
		in[TOP(x, y, w)] + in[BOT(x, y, w)]) / 4)

#define INTERPOLATE_X(in, x, y, w) \
	(((uint32_t)in[TL(x, y, w)] + in[BL(x, y, w)] + \
		in[TR(x, y, w)] + in[BR(x, y, w)]) / 4)

#define RED 0
#define GREEN 1
#define BLUE 2

struct cuda_vars {
	bayer_to_rgb_t kernel;
	dim3 threads_p_block;
	dim3 blocks_p_grid;

	int2 pos_r;
	int2 pos_gr;
	int2 pos_gb;
	int2 pos_b;

	uint8_t *d_bilinear;
	uint8_t *d_input;

	uint32_t width;
	uint32_t height;

	hipStream_t streams;

	uint8_t bpp;
};

/**
 * CUDA Kernel Device code for bayer to RGB
 *
 * Computes the Bilear Interpolation of missing coloured pixel from Bayer pattern.
 * Output is RGB.
 *
 * Each CUDA thread computes four pixels in a 2x2 square. Therefore no if
 * conditions are required, which slows the CUDA kernels massively.
 *
 * The first square starts with the pixel in position 1,1. Therefore the square
 * for each thread looks like this:
 *
 * B G
 * G R
 *
 * This approach saves one pixel lines at the edges of the image in contrast to
 * the first square at 2,2 with:
 *
 * R G
 * G B
 *
 * To support other formats than RGGB we also pass the position of each color
 * channel in the 2x2 block. In the above case we get B at 0,0, Gb at 1,0,
 * Gr at 0,1 and R at 1,1.
 */
__global__ void bayer_to_rgb(uint8_t *in, uint8_t *out, uint32_t imgw,
		uint32_t imgh, uint8_t bpp, int2 r, int2 gr, int2 gb, int2 b)
{
	int x = 2 * ((blockDim.x * blockIdx.x) + threadIdx.x) + 1;
	int y = 2 * ((blockDim.y * blockIdx.y) + threadIdx.y) + 1;
	int elemCols = imgw * bpp;

	if ((x + 2) < imgw && (x - 1) >= 0 && (y + 2) < imgh && (y - 1) >= 0) {
		/* Red */
		out[(y + r.y) * elemCols + (x + r.x) * bpp + RED] =
				PIX(in, x + r.x, y + r.y, imgw);
		out[(y + r.y) * elemCols + (x + r.x) * bpp + GREEN] =
				INTERPOLATE_HV(in, x + r.x, y + r.y, imgw);
		out[(y + r.y) * elemCols + (x + r.x) * bpp + BLUE] =
				INTERPOLATE_X(in, x + r.x, y + r.y, imgw);

		/* Green on a red line */
		out[(y + gr.y) * elemCols + (x + gr.x) * bpp + RED] =
				INTERPOLATE_H(in, x + gr.x, y + gr.y, imgw);
		out[(y + gr.y) * elemCols + (x + gr.x) * bpp + GREEN] =
				PIX(in, x + gr.x, y + gr.y, imgw);
		out[(y + gr.y) * elemCols + (x + gr.x) * bpp + BLUE] =
				INTERPOLATE_V(in, x + gr.x, y + gr.y, imgw);

		/* Green on a blue line */
		out[(y + gb.y) * elemCols + (x + gb.x) * bpp + RED] =
				INTERPOLATE_V(in, x + gb.x, y + gb.y, imgw);
		out[(y + gb.y) * elemCols + (x + gb.x) * bpp + GREEN] =
				PIX(in, x + gb.x, y + gb.y, imgw);
		out[(y + gb.y) * elemCols + (x + gb.x) * bpp + BLUE] =
				INTERPOLATE_H(in, x + gb.x, y + gb.y, imgw);

		/* Blue */
		out[(y + b.y) * elemCols + (x + b.x) * bpp + RED] =
				INTERPOLATE_X(in, x + b.x, y + b.y, imgw);
		out[(y + b.y) * elemCols + (x + b.x) * bpp + GREEN] =
				INTERPOLATE_HV(in, x + b.x, y + b.y, imgw);
		out[(y + b.y) * elemCols + (x + b.x) * bpp + BLUE] =
				PIX(in, x + b.x, y + b.y, imgw);

		if (bpp == 4) {
			out[y * elemCols + x * bpp + 3] = 255;
			out[y * elemCols + (x + 1) * bpp + 3] = 255;
			out[(y + 1) * elemCols + x * bpp + 3] = 255;
			out[(y + 1) * elemCols + (x + 1) * bpp + 3] = 255;
		}
	}
}

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int test_cuda(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  

  auto ret_val = hipMalloc(&d_x, N*sizeof(float)); 
	hipDeviceSynchronize();
  if (ret_val != hipSuccess) {
		fprintf(stderr, "test cudamalloc 1 %d, %s\n", 0,
				hipGetErrorString(ret_val));
		return ret_val;
	}
  ret_val = hipMalloc(&d_y, N*sizeof(float)); 
	hipDeviceSynchronize();
  if (ret_val != hipSuccess) {
		fprintf(stderr, "test cudamalloc 2 %d, %s\n", 0,
				hipGetErrorString(ret_val));
		return ret_val;
	}

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }


  ret_val = hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
  if (ret_val != hipSuccess) {
		fprintf(stderr, "test Host to Device %d, %s\n", 0,
				hipGetErrorString(ret_val));
		return ret_val;
	}

  ret_val = hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
  if (ret_val != hipSuccess) {
		fprintf(stderr, "test2 Host to Device %d, %s\n", 0,
				hipGetErrorString(ret_val));
		return ret_val;
	}

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}

hipError_t bayer2rgb_process(struct cuda_vars *gpu_vars, const void *p,
		uint8_t **output, hipStream_t *stream, bool get_dev_ptr)
{
	hipError_t ret_val;

	if (gpu_vars == NULL)
		return hipErrorNotInitialized;


	ret_val = hipMemcpy(gpu_vars->d_input, p, gpu_vars->width*gpu_vars->height * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (ret_val != hipSuccess) {
		fprintf(stderr, "Host to Device %d, %s\n", 0,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	gpu_vars->kernel<<<gpu_vars->blocks_p_grid,
			gpu_vars->threads_p_block, 0,
			gpu_vars->streams
		>>>(gpu_vars->d_input,
			gpu_vars->d_bilinear,
			gpu_vars->width, gpu_vars->height, gpu_vars->bpp,
			gpu_vars->pos_r, gpu_vars->pos_gr,
			gpu_vars->pos_gb, gpu_vars->pos_b);

	if (get_dev_ptr) {
		*output = (uint8_t *)gpu_vars->d_bilinear;
	} else {
	}

	*stream = gpu_vars->streams;


	return hipSuccess;
}

hipError_t alloc_create_cuda_data(struct cuda_vars *gpu_vars)
{
	hipError_t ret_val = hipSuccess;

	ret_val = hipMalloc(&gpu_vars->d_input, gpu_vars->width * gpu_vars->height * sizeof(uint8_t));
	if (ret_val != hipSuccess) {
		fprintf(stderr, "hipMalloc d_input %d, %s\n", 0,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	ret_val = hipMalloc(&gpu_vars->d_bilinear, gpu_vars->width *
			gpu_vars->height * gpu_vars->bpp * sizeof(uint8_t));
	if (ret_val != hipSuccess) {
		fprintf(stderr, "hipMalloc d_bilinear %d, %s\n", 0,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	ret_val = hipStreamCreate(&gpu_vars->streams);
	if (ret_val != hipSuccess) {
		fprintf(stderr, "hipStreamCreate %d, %s\n", 0,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	return ret_val;
}

hipError_t bayer2rgb_init(struct cuda_vars **gpu_vars_p, uint32_t width,
		uint32_t height, uint8_t bpp, bool thermal)
{
	struct cuda_vars *gpu_vars;
	bayer_to_rgb_t ir_kernel;
	hipError_t ret_val;
	int i;

	if (gpu_vars_p == NULL)
		return hipErrorNotInitialized;

	gpu_vars = (cuda_vars *) new(struct cuda_vars);
	if (!gpu_vars)
		return hipErrorOutOfMemory;

	gpu_vars->width = width;
	gpu_vars->height = height;
	gpu_vars->bpp = bpp;
	gpu_vars->kernel = bayer_to_rgb;

	gpu_vars->pos_r = make_int2(1, 1);
	gpu_vars->pos_gr = make_int2(0, 1);
	gpu_vars->pos_gb = make_int2(1, 0);
	gpu_vars->pos_b = make_int2(0, 0);

	ret_val = alloc_create_cuda_data(gpu_vars);
	if (ret_val != hipSuccess)
		goto cleanup;

	gpu_vars->threads_p_block = dim3(32, 32);
	gpu_vars->blocks_p_grid.x = (gpu_vars->width / 2 +
			gpu_vars->threads_p_block.x - 1) /
			gpu_vars->threads_p_block.x;
	gpu_vars->blocks_p_grid.y = (gpu_vars->height / 2 +
			gpu_vars->threads_p_block.y - 1) /
			gpu_vars->threads_p_block.y;

	*gpu_vars_p = gpu_vars;

	return hipSuccess;

cleanup:
	bayer2rgb_free(gpu_vars);

	return ret_val;
}

void free_cuda_data(struct cuda_vars *gpu_vars)
{
	if (gpu_vars->d_input)
		hipFree(gpu_vars->d_input);
	if (gpu_vars->d_bilinear)
		hipFree(gpu_vars->d_bilinear);
	hipStreamDestroy(gpu_vars->streams);
}

hipError_t bayer2rgb_free(struct cuda_vars *gpu_vars)
{
	free_cuda_data(gpu_vars);

	free(gpu_vars);

	return hipSuccess;
}



// From here: https://stackoverflow.com/questions/61457243/problem-of-converting-bgr-to-yuv420p-with-cuda
__host__ __device__ unsigned char rgb2y(int R, int G, int B){
  int Y = ((66 * R + 129 * G + 25 * B + 128) >> 8) + 16;
  return (unsigned char)((Y<0)? 0 : ((Y > 255) ? 255 : Y));}
__host__ __device__ int rgb2u(int R, int G, int B){
  int U = ((-38 * R - 74 * G + 112 * B + 128) >> 8) + 128;
  return (unsigned char)((U<0)? 0 : ((U > 255) ? 255 : U));}
__host__ __device__ int rgb2v(int R, int G, int B){
  int V = ((112 * R - 94 * G - 18 * B + 128) >> 8) + 128;
  return (unsigned char)((V<0)? 0 : ((V > 255) ? 255 : V));}

//kernel function to convert rgb to yuv420p
__global__ void rgb2yuv420p(uint8_t *d_in, uint8_t *d_out,
                               uint imgheight, uint imgwidth)
{

    int col_num = blockIdx.x*blockDim.x+threadIdx.x;
    int row_num = blockIdx.y*blockDim.y+threadIdx.y;

    if ((row_num < imgheight) && (col_num < imgwidth))
    {
//        uint32_t a = *((uint32_t *)&dinput[global_offset*3]);
        int global_offset = row_num * imgwidth * 3 + col_num * 3;

        int r,g,b;
        
        r = int(d_in[global_offset + RED]);
        g = int(d_in[global_offset + GREEN]);
        b = int(d_in[global_offset + BLUE]);

        d_out[row_num * imgwidth + col_num] = rgb2y(r,g,b);
		// https://stackoverflow.com/questions/27822017/planar-yuv420-data-layout

        if(((threadIdx.x & 1) == 0)  && ((threadIdx.y & 1) == 0)){ // 1 = 0001
			int u_offset = imgwidth*imgheight+((row_num>>1)*(imgwidth>>1))+(col_num>>1);
            d_out[u_offset] = rgb2u(r,g,b);
            int v_offset = u_offset+((imgheight>>1)*(imgwidth>>1));
            d_out[v_offset] = rgb2v(r,g,b);
        }
    }
}

hipError_t rgb2yuv420p_process(uint8_t *d_in, uint8_t *d_out,
                               uint imgheight, uint imgwidth)
{
	hipError_t ret_val;
	dim3 threadsPerBlock(32, 32);
	dim3 blocksPerGrid((imgwidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
					(imgheight + threadsPerBlock.y - 1) / threadsPerBlock.y);

	//run rgb->yuv420p kernel function
	rgb2yuv420p<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, imgheight, imgwidth);

	return hipSuccess;
}